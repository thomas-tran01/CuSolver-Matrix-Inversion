#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <iomanip> 

#define MATRIX_DIM 5000
#define ITER 10

void createMatrix(float* h_matrix)
{
    for(int i = 0; i < MATRIX_DIM * MATRIX_DIM; i++)
    {
        h_matrix[i] = (rand()) % 100;
    }
}

void setIdentityMatrix(float* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            matrix[i*size+j] = (i == j) ? 1.0f : 0.0f;
        }
    }
}


void printMatrix(const float* matrix) {
    for (int i = 0; i < MATRIX_DIM; ++i) {
        for (int j = 0; j < MATRIX_DIM; ++j) {
            std::cout << matrix[i * MATRIX_DIM + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main()
{
    const int matrixSize = MATRIX_DIM * MATRIX_DIM * sizeof(float);
    float res[ITER];
	float *curr = res;
    for (int i = 0; i < ITER; ++i)
    {
        //Host
        float h_matrix[MATRIX_DIM*MATRIX_DIM];
        float h_invMatrix[MATRIX_DIM*MATRIX_DIM];
        createMatrix(h_matrix);
        //printMatrix(h_matrix);
        
        // Device
        float* d_matrix;
        float* d_invMatrix;
        float* d_identity;
        int* d_info;
        float* d_work;
        int* d_ipiv;
        int workspace_size = 0;
        hipMalloc(&d_matrix, matrixSize);
        hipMalloc(&d_invMatrix, matrixSize);
        hipMalloc(&d_identity, matrixSize);
        hipMalloc(&d_info, sizeof(int));
        hipMalloc(&d_work, workspace_size * sizeof(float));
        hipMalloc(&d_ipiv, MATRIX_DIM * sizeof(int));
        hipMemcpy(d_matrix, h_matrix, matrixSize, hipMemcpyHostToDevice);


        hipEvent_t start, stop;
        float computation_time;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start,0);
        

        hipsolverHandle_t cussolverHandle;
        hipsolverDnCreate(&cussolverHandle);

        hipsolverDnSgetrf_bufferSize(cussolverHandle, MATRIX_DIM, MATRIX_DIM, d_matrix, MATRIX_DIM, &workspace_size);

        // LU factorization
        hipsolverDnSgetrf(cussolverHandle, MATRIX_DIM, MATRIX_DIM, d_matrix, MATRIX_DIM, d_work, d_ipiv, d_info);
        hipDeviceSynchronize();

        // Identity matrix
        float h_identity[MATRIX_DIM*MATRIX_DIM];
        setIdentityMatrix(h_identity, MATRIX_DIM);

        hipMemcpy(d_identity, h_identity, matrixSize, hipMemcpyHostToDevice);

        // Invert the matrix
        hipsolverDnSgetrs(cussolverHandle, HIPBLAS_OP_N, MATRIX_DIM, MATRIX_DIM, d_matrix, MATRIX_DIM, d_ipiv, d_identity, MATRIX_DIM, d_info);
        hipDeviceSynchronize();
        hipMemcpy(h_invMatrix, d_identity, matrixSize, hipMemcpyDeviceToHost);

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&computation_time, start, stop);

        *curr = computation_time;
        ++curr;

        std::cout<< std::fixed << std::setprecision(5);
        std::cout<< "Computation took " << computation_time << "ms" << std::endl;
        //std::cout<< "Inverted Matrix:"<< std::endl;
        //std::cout<< h_invMatrix << std::endl;
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_matrix);
        hipFree(d_invMatrix);
        hipFree(d_identity);
        hipFree(d_info);
        hipFree(d_work);
        hipFree(d_ipiv);
        hipsolverDnDestroy(cussolverHandle);

    }

    float total = 0.0;

	for (int x = 0; x < ITER; ++x)
	{
		total += res[x];
	}
	std::cout << "avg Cuda Time: " << total/ITER << "ms\n";\

}
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MATRIX_DIM 3
__global__ void initializeMatrix(float* d_matrix, int matrixDim, int seed)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int index = idy * matrixDim + idx;

    // Initialize matrix with random int from 0-100
    if (idx < matrixDim && idy < matrixDim)
    {
        hiprandState state;
        hiprand_init(seed, index, 0, &state);
        d_matrix[index] = static_cast<float>(hiprand(&state) % 100);
    }
}


void printMatrix(const float* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            std::cout << matrix[i * size + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main()
{
    const int matrixSize = MATRIX_DIM * MATRIX_DIM * sizeof(float);
    float h_matrix[MATRIX_DIM*MATRIX_DIM];
    float h_invMatrix[MATRIX_DIM*MATRIX_DIM];
    
    float* d_matrix;
    hipMalloc(&d_matrix, matrixSize);

    dim3 threadsPerBlock(MATRIX_DIM, MATRIX_DIM);
    dim3 numBlocks(1, 1);
    initializeMatrix<<<numBlocks, threadsPerBlock>>>(d_matrix, MATRIX_DIM, time(NULL));
    hipMemcpy(h_matrix, d_matrix, matrixSize, hipMemcpyDeviceToHost);

    std::cout << "Current matrix:" << std::endl;
    printMatrix((float*)h_matrix, MATRIX_DIM);
}
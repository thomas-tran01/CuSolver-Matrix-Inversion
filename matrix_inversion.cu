#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsolver.h>

#define MATRIX_DIM 300

__global__ void initializeMatrix(float* d_matrix, int matrixDim, int seed)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int index = idy * matrixDim + idx;

    // Initialize matrix with random int from 0-100
    if (idx < matrixDim && idy < matrixDim)
    {
        hiprandState state;
        hiprand_init(seed, index, 0, &state);
        d_matrix[index] = static_cast<float>(hiprand(&state) % 100);
    }
}

__global__ void setIdentityMatrix(float* matrix, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n) {
        int row = idx / n;
        int col = idx % n;
        matrix[idx] = (row == col) ? 1.0f : 0.0f;
    }
}


void printMatrix(const float* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            std::cout << matrix[i * size + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main()
{
    const int matrixSize = MATRIX_DIM * MATRIX_DIM * sizeof(float);

    //Host
    float h_matrix[MATRIX_DIM*MATRIX_DIM];
    float h_invMatrix[MATRIX_DIM*MATRIX_DIM];
    
    // Device
    float* d_matrix;
    float* d_invMatrix;
    int* d_info;
    hipMalloc(&d_matrix, matrixSize);
    hipMalloc(&d_invMatrix, matrixSize);
    hipMalloc(&d_info, sizeof(int));

    int threadsPerBlock = 16; 
    dim3 blockSize(threadsPerBlock, threadsPerBlock);
    dim3 gridSize((MATRIX_DIM + threadsPerBlock - 1) / threadsPerBlock, 
                (MATRIX_DIM + threadsPerBlock - 1) / threadsPerBlock);


    hipEvent_t start, stop;
    float computation_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    initializeMatrix<<<gridSize, blockSize>>>(d_matrix, MATRIX_DIM, time(NULL));
    hipMemcpy(h_matrix, d_matrix, matrixSize, hipMemcpyDeviceToHost);

    std::cout << "Current matrix:" << std::endl;
    //printMatrix((float*)h_matrix, MATRIX_DIM);

    hipsolverHandle_t cussolverHandle;
    hipsolverDnCreate(&cussolverHandle);
    
    // Workspace size query
    int workspace_size = 0;
    hipsolverDnSgetrf_bufferSize(cussolverHandle, MATRIX_DIM, MATRIX_DIM, d_matrix, MATRIX_DIM, &workspace_size);
    size_t free, total;
    hipMemGetInfo(&free, &total);
    std::cout << "GPU memory: " << free / (1024*1024) << "MB free of " << total / (1024*1024) << "MB total" << std::endl;
    std::cout << "Workspace size:" << workspace_size << std::endl;
    std::cout << "Workspace size: " << workspace_size * sizeof(float) / (1024*1024) << " MB" << std::endl;

    float* d_work;
    hipMalloc(&d_work, workspace_size * sizeof(float));

    // LU factorization
    int* d_ipiv;
    hipMalloc(&d_ipiv, MATRIX_DIM * sizeof(int));
    hipsolverDnSgetrf(cussolverHandle, MATRIX_DIM, MATRIX_DIM, d_matrix, MATRIX_DIM, d_work, d_ipiv, d_info);

    // Copy LU matrix back to host
    hipMemcpy(h_matrix, d_matrix, matrixSize, hipMemcpyDeviceToHost);
    std::cout << "LU matrix:" << std::endl;
    //printMatrix((float*)h_matrix, MATRIX_DIM);

    int h_info[1];
    hipMemcpy(h_info,d_info,1*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < 1; ++i)
    {
        std::cout << "h_info: " << i << " = " << h_info[i] << std::endl;
    }

    // Identity matrix
    float* d_identity;
    hipMalloc(&d_identity, matrixSize);
    hipMemset(d_identity, 0, matrixSize);
    int threadsPerBlock1D = 256;
    int numBlocks1D = (MATRIX_DIM * MATRIX_DIM + threadsPerBlock1D - 1) / threadsPerBlock1D;
    setIdentityMatrix<<<numBlocks1D, threadsPerBlock1D>>>(d_identity, MATRIX_DIM);

    // Invert the matrix
    hipsolverDnSgetrs(cussolverHandle, HIPBLAS_OP_N, MATRIX_DIM, MATRIX_DIM, d_matrix, MATRIX_DIM, d_ipiv, d_identity, MATRIX_DIM, d_info);

    // Copy inverted matrix back to host
    hipMemcpy(h_invMatrix, d_identity, matrixSize, hipMemcpyDeviceToHost);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&computation_time, start, stop);

    printf("Inverted Matrix:\n");
    //printMatrix(h_invMatrix, MATRIX_DIM);
    printf("Computation took %.10fms\n", computation_time);

    hipEventDestroy(start);
    hipEventDestroy(stop);


    hipFree(d_matrix);
    hipFree(d_invMatrix);
    hipFree(d_identity);
    hipFree(d_info);
    hipFree(d_work);
    hipFree(d_ipiv);
    hipsolverDnDestroy(cussolverHandle);

}
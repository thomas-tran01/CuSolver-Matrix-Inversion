#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <iomanip> 
#include <fstream>
#include <chrono>
#include <sstream>
#define MATRIX_DIM 1000
#define ITER 100

void createMatrix(float* h_matrix)
{
    for(int i = 0; i < MATRIX_DIM * MATRIX_DIM; i++)
    {
        h_matrix[i] = (rand()) % 100;
    }
}

void setIdentityMatrix(float* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            matrix[i*size+j] = (i == j) ? 1.0f : 0.0f;
        }
    }
}


void printMatrix(const float* matrix) {
    for (int i = 0; i < MATRIX_DIM; ++i) {
        for (int j = 0; j < MATRIX_DIM; ++j) {
            std::cout << matrix[i * MATRIX_DIM + j] << " ";
        }
        std::cout << std::endl;
    }
}

std::string getLogFileName() {
    auto now = std::chrono::system_clock::now();
    auto in_time_t = std::chrono::system_clock::to_time_t(now);

    std::stringstream ss;
    ss << std::put_time(std::localtime(&in_time_t), "%Y%m%d_%H%M%S");
    std::string filename = "log_" + ss.str() + ".csv";
    return filename;
}


int main()
{
    std::ofstream logFile(getLogFileName());
    const int matrixSize = MATRIX_DIM * MATRIX_DIM * sizeof(float);
    float res[ITER];
	float *curr = res;
    for (int i = 0; i < ITER; ++i)
    {
        //Host
        float* h_matrix = (float*)malloc(MATRIX_DIM * MATRIX_DIM * sizeof(float));
        float* h_invMatrix = (float*)malloc(MATRIX_DIM * MATRIX_DIM * sizeof(float));
        createMatrix(h_matrix);
        //printMatrix(h_matrix);
        
        // Device
        float* d_matrix;
        float* d_invMatrix;
        float* d_identity;
        int* d_info;
        float* d_work;
        int* d_ipiv;
        int workspace_size = 0;
        hipMalloc(&d_matrix, matrixSize);
        hipMalloc(&d_invMatrix, matrixSize);
        hipMalloc(&d_identity, matrixSize);
        hipMalloc(&d_info, sizeof(int));
        hipMalloc(&d_ipiv, MATRIX_DIM * sizeof(int));
        hipMemcpy(d_matrix, h_matrix, matrixSize, hipMemcpyHostToDevice);


        hipEvent_t start, stop;
        float computation_time;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start,0);
        

        hipsolverHandle_t cussolverHandle;
        hipsolverDnCreate(&cussolverHandle);

        hipsolverDnSgetrf_bufferSize(cussolverHandle, MATRIX_DIM, MATRIX_DIM, d_matrix, MATRIX_DIM, &workspace_size);
        hipMalloc(&d_work, workspace_size * sizeof(float));

        // LU factorization
        hipsolverDnSgetrf(cussolverHandle, MATRIX_DIM, MATRIX_DIM, d_matrix, MATRIX_DIM, d_work, d_ipiv, d_info);
        hipDeviceSynchronize();

        // Identity matrix
        float* h_identity = (float*)malloc(MATRIX_DIM * MATRIX_DIM * sizeof(float));
        setIdentityMatrix(h_identity, MATRIX_DIM);

        hipMemcpy(d_identity, h_identity, matrixSize, hipMemcpyHostToDevice);

        // Invert the matrix
        hipsolverDnSgetrs(cussolverHandle, HIPBLAS_OP_N, MATRIX_DIM, MATRIX_DIM, d_matrix, MATRIX_DIM, d_ipiv, d_identity, MATRIX_DIM, d_info);
        hipDeviceSynchronize();
        hipMemcpy(h_invMatrix, d_identity, matrixSize, hipMemcpyDeviceToHost);

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&computation_time, start, stop);

        *curr = computation_time;
        ++curr;

        std::cout<< std::fixed << std::setprecision(5);
        std::cout<< "Computation took " << computation_time << "ms" << std::endl;
        logFile << computation_time << "\n";
        //std::cout<< "Inverted Matrix:"<< std::endl;
        //std::cout<< h_invMatrix << std::endl;
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_matrix);
        hipFree(d_invMatrix);
        hipFree(d_identity);
        hipFree(d_info);
        hipFree(d_work);
        hipFree(d_ipiv);
        hipsolverDnDestroy(cussolverHandle);

    }

    float total = 0.0;

	for (int x = 0; x < ITER; ++x)
	{
		total += res[x];
	}
	std::cout << "avg Cuda Time: " << total/ITER << "ms\n";\

}
